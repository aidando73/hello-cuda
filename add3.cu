#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int blockIdx = blockIdx.x;
  int stride = blockDim.x;
  int i = blockIdx * stride + index;
  y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  // Run kernel on 1M elements on the CPU
  // Get start time
  clock_t start = clock();

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Get end time and compute duration
  clock_t end = clock();
  double duration = ((double)(end - start)) / CLOCKS_PER_SEC * 1000000; // Convert to microseconds
  std::cout << "Time taken: " << duration << " us" << std::endl;

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}